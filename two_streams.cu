#include <iostream>
#include <hip/hip_runtime.h>

__global__ void printHello(int threadId) {
    for (int i = 0; i < 100000; ++i) {
        printf("Hello from thread %d\n", threadId);
    }
}

int main() {
    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);

    // Launch kernel in stream 1
    printHello<<<1, 1, 0, stream1>>>(1);

    // Launch kernel in stream 2
    printHello<<<1, 1, 0, stream2>>>(2);

    // Synchronize streams
    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);

    // Destroy streams
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);

    return 0;
}
