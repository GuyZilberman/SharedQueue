#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in 
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <stdlib.h>
#include <getopt.h>
#include <memory.h>
#include <stdio.h>
#include <math.h>
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
 #include <thread>

using namespace std;

#include "gdrapi.h"
#include "gdrcopy_common.hpp"

using namespace gdrcopy::test;

void pp_cpu_thread(gdr_mh_t mh, uint32_t *d_buf, uint32_t *h_buf, uint32_t num_iters){
        uint32_t i = 1;
        // Wait for pp_kernel to be ready before starting the time measurement.
        while (READ_ONCE(*h_buf) != i);
        LB();

        // Restart the timer for measurement.
        while (i < num_iters) {
            gdr_copy_to_mapping(mh, d_buf, &i, sizeof(d_buf));
            SB();

            ++i;

            while (READ_ONCE(*h_buf) != i);
            LB();
        }
}

__global__ void pp_kernel(uint32_t *d_buf, uint32_t *h_buf, uint32_t num_iters)
{
    uint32_t i = 1;
    WRITE_ONCE(*h_buf, i);
    __threadfence_block();
    while (i < num_iters) {
        while (READ_ONCE(*d_buf) != i) ;
        __threadfence_block();

        ++i;
        WRITE_ONCE(*h_buf, i);
        __threadfence_block();
    }
}

static int dev_id = 0;
static uint32_t num_iters = 5;

int main(int argc, char *argv[])
{
    uint32_t *d_buf = NULL;
    uint32_t *h_buf = NULL;

    hipDeviceptr_t d_buf_cuptr;
    hipDeviceptr_t h_buf_cuptr;

    gpu_mem_handle_t mhandle;

    // GUY: Initialize the CUDA driver API
    ASSERTDRV(hipInit(0));

    // GUY: Start of device selection stuff
    int n_devices = 0;
    ASSERTDRV(hipGetDeviceCount(&n_devices));

    hipDevice_t dev;
    for (int n=0; n<n_devices; ++n) {
        
        char dev_name[256];
        int dev_pci_domain_id;
        int dev_pci_bus_id;
        int dev_pci_device_id;

        ASSERTDRV(hipDeviceGet(&dev, n));
        ASSERTDRV(hipDeviceGetName(dev_name, sizeof(dev_name) / sizeof(dev_name[0]), dev));
        ASSERTDRV(hipDeviceGetAttribute(&dev_pci_domain_id, hipDeviceAttributePciDomainId, dev));
        ASSERTDRV(hipDeviceGetAttribute(&dev_pci_bus_id, hipDeviceAttributePciBusId, dev));
        ASSERTDRV(hipDeviceGetAttribute(&dev_pci_device_id, hipDeviceAttributePciDeviceId, dev));

        cout  << "GPU id:" << n << "; name: " << dev_name 
              << "; Bus id: "
              << std::hex 
              << std::setfill('0') << std::setw(4) << dev_pci_domain_id
              << ":" << std::setfill('0') << std::setw(2) << dev_pci_bus_id
              << ":" << std::setfill('0') << std::setw(2) << dev_pci_device_id
              << std::dec
              << endl;
    }
    cout << "selecting device " << dev_id << endl;
    ASSERTDRV(hipDeviceGet(&dev, dev_id));

    hipCtx_t dev_ctx;
    ASSERTDRV(hipDevicePrimaryCtxRetain(&dev_ctx, dev));
    ASSERTDRV(hipCtxSetCurrent(dev_ctx));

    // Check that the device supports GDR
    ASSERT_EQ(check_gdr_support(dev), true);
    // GUY: End of device selection stuff

    ASSERTDRV(gpu_mem_alloc(&mhandle, sizeof(*d_buf), true, true));
    d_buf_cuptr = mhandle.ptr;
    cout << "device ptr: 0x" << hex << d_buf_cuptr << dec << endl;

    // set d_buf_cuptr's value to 0
    ASSERTDRV(hipMemsetD8(d_buf_cuptr, 0, sizeof(*d_buf)));

    ASSERTDRV(hipHostAlloc((void **)&h_buf, sizeof(*h_buf), hipHostMallocPortable | hipHostMallocMapped));
    ASSERT_NEQ(h_buf, (void*)0);
    ASSERTDRV(hipHostGetDevicePointer(&h_buf_cuptr, h_buf, 0));
    memset(h_buf, 0, sizeof(*h_buf));

    // called to open a handle to the GPUDirect RDMA driver
    gdr_t g = gdr_open_safe();

    gdr_mh_t mh;
    BEGIN_CHECK {
        // tokens are optional in CUDA 6.0
        // Create a peer-to-peer mapping of the device memory buffer, returning an opaque handle.
        ASSERT_EQ(gdr_pin_buffer(g, d_buf_cuptr, sizeof(*d_buf), 0, 0, &mh), 0);
        ASSERT_NEQ(mh, null_mh);

        void *map_d_ptr  = NULL;
        ASSERT_EQ(gdr_map(g, mh, &map_d_ptr, sizeof(*d_buf)), 0);
        cout << "map_d_ptr: " << map_d_ptr << endl;

        gdr_info_t info;
        ASSERT_EQ(gdr_get_info(g, mh, &info), 0);
        cout << "info.va: " << hex << info.va << dec << endl;
        cout << "info.mapped_size: " << info.mapped_size << endl;
        cout << "info.page_size: " << info.page_size << endl;
        cout << "info.mapped: " << info.mapped << endl;
        cout << "info.wc_mapping: " << info.wc_mapping << endl;

        // remember that mappings start on a 64KB boundary, so let's
        // calculate the offset from the head of the mapping to the
        // beginning of the buffer
        int off = info.va - d_buf_cuptr;
        cout << "page offset: " << off << endl;

        d_buf = (uint32_t *)((uintptr_t)map_d_ptr + off);
        cout << "user-space pointer: " << d_buf << endl;

        cout << "CPU does gdr_copy_to_mapping and GPU writes back via hipHostAlloc'd buffer." << endl;
        cout << "Running " << num_iters << " iterations with data size " << sizeof(*d_buf) << " bytes." << endl;

        pp_kernel<<< 1, 1 >>>((uint32_t *)d_buf_cuptr, (uint32_t *)h_buf_cuptr, num_iters);

        // Catching any potential errors. hipErrorNotReady means pp_kernel
        // is running. We expect to see this status instead of hipSuccess
        // because pp_kernel must wait for signal from CPU, which occurs after
        // this line.
        ASSERT_EQ(hipStreamQuery(0), hipErrorNotReady);

        // Launch a server thread
        std::thread server_thread(pp_cpu_thread, mh, d_buf, h_buf, num_iters);
        server_thread.detach();

        ASSERTDRV(hipStreamSynchronize(0));

        cout << "unmapping buffer" << endl;
        ASSERT_EQ(gdr_unmap(g, mh, map_d_ptr, sizeof(*d_buf)), 0);

        cout << "unpinning buffer" << endl;
        ASSERT_EQ(gdr_unpin_buffer(g, mh), 0);
    } END_CHECK;

    cout << "closing gdrdrv" << endl;
    ASSERT_EQ(gdr_close(g), 0);

    ASSERTDRV(hipHostFree(h_buf));
    ASSERTDRV(gpu_mem_free(&mhandle));

    return 0;
}
