#include "hip/hip_runtime.h"
#include "common.cuh"
#include "shared_queue.cuh"
#include "/etc/pliops/store_lib_expo.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>


#include "gdrapi.h"
#include "gdrcopy_common.hpp"
#include "gdr_gpu_memalloc.cuh"

#define NO_OPTIONS 0
#define NUM_ITERATIONS QUEUE_SIZE
#define READ_START_ID NUM_ITERATIONS
#define READ_END_ID 2*NUM_ITERATIONS-1

__device__
void InitData(int* arr, size_t size, int idx) {
    // Set the first value in the vector to idx
    arr[0] = idx;
}

__global__
void client_thread_func(LockFreeQueue<RequestMessage> *submission_queue, LockFreeQueue<ResponseMessage> *completion_queue, const int num_iterations){
    uint idx = 0, request_id = 0;
    AnswerType answer = AnswerType::NONE;

    // Send write requests
    while (idx < num_iterations){ 
        // Perform IO request
        printf("submission_queue: before push idx is %d\n", idx);
        RequestMessage req_msg;
        req_msg.cmd = CommandType::WRITE;
        req_msg.request_id = request_id++;
        InitData(req_msg.data, 256, idx);
        req_msg.key = idx++;
        while (!submission_queue->push(req_msg)); // Busy-wait until the value is pushed successfully

        // Immediately wait for a response
        ResponseMessage res_msg;
        while (!completion_queue->pop(res_msg)); // Busy-wait for a command to be available
        // TODO guy check about this: Optional: backoff strategy to reduce CPU usage
        answer = res_msg.answer;
        printf("Client: Received from completion queue: %d\n", (int)answer);
    }

    // Send read requests
    idx = 0;
    while (idx < num_iterations){
        RequestMessage req_msg;
        req_msg.cmd = CommandType::READ;
        req_msg.request_id = request_id++;
        req_msg.key = idx++;

        while (!submission_queue->push(req_msg)); // Busy-wait until the value is pushed successfully

        // Immediately wait for a response
        ResponseMessage res_msg;
        while (!completion_queue->pop(res_msg)); // Busy-wait for a command to be available
        // TODO guy check about this: Optional: backoff strategy to reduce CPU usage
        answer = res_msg.answer;
        printf("Client: Received from completion queue: %d\n", (int)answer);
    }

    while (answer != AnswerType::EXIT)
    {
        // Send exit request
        RequestMessage req_msg_exit;
        req_msg_exit.cmd = CommandType::EXIT;
        req_msg_exit.request_id = ++request_id;
        while (!submission_queue->push(req_msg_exit)); // Busy-wait until the value is pushed successfully

        // Immediately wait for a response
        ResponseMessage res_msg;
        while (!completion_queue->pop(res_msg)); // Busy-wait for a command to be available
        answer = res_msg.answer;
        printf("Client: Received from completion queue: %d\n", (int)answer);
        printf("Client: data[0] from completion queue: %d\n", res_msg.data[0]);
    }
}

//void server_func(LockFreeQueue<RequestMessage> *submission_queue, LockFreeQueue<ResponseMessage> *completion_queue, sem_t* p_server_semaphore, PLIOPS_DB_t plio_handle ){
void server_func(LockFreeQueue<RequestMessage> *submission_queue, LockFreeQueue<ResponseMessage> *completion_queue, PLIOPS_DB_t plio_handle ){
    uint actual_object_size = 0;
    int ret = 0;
    RequestMessage req_msg; // TODO guy move this into the while loop
    CommandType command = CommandType::NONE;
    int idx = 0;
    // Signal that initialization is done
    // sem_post(p_server_semaphore);
    
    while (command != CommandType::EXIT) {
        ResponseMessage res_msg;
        while (!submission_queue->pop(req_msg)); // Busy-wait for a value to be available
        command = req_msg.cmd;
        res_msg.request_id = req_msg.request_id;

            if (req_msg.cmd == CommandType::EXIT){
                res_msg.answer = AnswerType::EXIT;
            }
            else if (req_msg.cmd == CommandType::WRITE)
            {
                std::cout << "Received: " << req_msg.data[0] << std::endl;
                std::cout << req_msg.request_id << ": Calling PLIOPS_Put! Value: "  << req_msg.data[0] << std::endl;
                ret = PLIOPS_Put(plio_handle, &req_msg.key, sizeof(req_msg.key), &req_msg.data, sizeof(req_msg.data), NO_OPTIONS); //TODO guy look into options
                if (ret != 0) {
                    printf("PLIOPS_Put Failed ret=%d\n", ret);
                    res_msg.answer = AnswerType::FAIL;
                    res_msg.error = ret;
                }
                else
                    res_msg.answer = AnswerType::SUCCESS; // TODO guy - res_msg.answer = SUCCESS;
                //std::cout << "Finished PLIOPS_Put!" << std::endl; 
            }
            else if (req_msg.cmd == CommandType::READ)
            {
                //std::cout << "Calling PLIOPS_Get!" << std::endl;
                ret = PLIOPS_Get(plio_handle, &req_msg.key, sizeof(req_msg.key), &res_msg.data, sizeof(res_msg.data), &actual_object_size);
                if (ret != 0) {
                    printf("PLIOPS_Get Failed ret=%d\n", ret);
                    res_msg.answer = AnswerType::FAIL;
                    res_msg.error = ret;
                }
                else
                    res_msg.answer = AnswerType::SUCCESS; // TODO guy - res_msg.answer = SUCCESS;
                //std::cout << "Finished PLIOPS_Get!" << std::endl; 
                //std::cout << req_msg.request_id << ": Called PLIOPS_Get! Value: " << res_msg.data[0] << std::endl;
            }
            else
            {
                //std::cout << "Cannot perform command " << (int)req_msg.cmd << std::endl;
                res_msg.answer = AnswerType::FAIL;
                //TODO add: res_msg.error = ???;
            }
        std::cout << idx << ": Before sending response message" << std::endl;
        while (!completion_queue->push(res_msg)); // Busy-wait until the value is pushed successfully
        std::cout << idx++ << ": After sending response message" << std::endl;
        //std::cout << "Server sent confirmation message with the answer: " << (int)res_msg.answer << std::endl;

    }
    
}

bool storelib_init(PLIOPS_IDENTIFY_t& identify, PLIOPS_DB_t& plio_handle){
    PLIOPS_DB_OPEN_OPTIONS_t db_open_options; //TODO guy check what each flag in the option does
    db_open_options.createIfMissing = 1;
    db_open_options.tailSizeInBytes = 0;
    // TODO guy ask ido: db_open_options.errorIfExists = ???

    int ret = PLIOPS_OpenDB(identify, &db_open_options, 0, &plio_handle);
    if (ret != 0) {
        printf("PLIOPS_OpenDB Failed ret=%d\n", ret);
        return false;
    }
    std::cout << "Finished PLIOPS_OpenDB!" << std::endl;   
    return true;
}

bool storelib_deinit(PLIOPS_IDENTIFY_t& identify, PLIOPS_DB_t& plio_handle){
    int ret = PLIOPS_CloseDB(plio_handle);
    if (ret != 0) {
        printf("PLIOPS_CloseDB Failed ret=%d\n", ret);
        return false;
    }
    std::cout << "Finished PLIOPS_CloseDB!" << std::endl;       

    ret = PLIOPS_DeleteDB(identify, 0);
    if (ret != 0) {
        printf("PLIOPS_DeleteDB Failed ret=%d\n", ret);
        return false;
    }
    std::cout << "Finished PLIOPS_DeleteDB!" <<std::endl;  
    return true;
}

bool process_requests(PLIOPS_DB_t& plio_handle){
    // sem_t server_semaphore;
    // sem_init(&server_semaphore, 0, 0); // 0 - shared between threads of a process, 0 - initial value

    LockFreeQueue<RequestMessage>* h_sq_p;
    LockFreeQueue<ResponseMessage>* h_cq_p;
    hipDeviceptr_t d_cq_p; // NEW
    GPUMemoryManager *gpu_mm = new GPUMemoryManager(); // NEW

    // Two queues - Allocate memory that is shared by the CPU and the GPU
    CUDA_ERRCHECK(hipHostAlloc((void **)&h_sq_p, sizeof(LockFreeQueue<RequestMessage>), hipHostMallocMapped));
	//CUDA_ERRCHECK(hipHostAlloc((void **)&h_cq_p, sizeof(LockFreeQueue<ResponseMessage>), hipHostMallocMapped));
    cudaGPUMemAlloc<LockFreeQueue<ResponseMessage>>(gpu_mm, &h_cq_p, d_cq_p);


    new (h_sq_p) LockFreeQueue<RequestMessage>();
    new (h_cq_p) LockFreeQueue<ResponseMessage>();

    // Launch a server thread
    std::thread server_thread(server_func, h_sq_p, h_cq_p, plio_handle);
    server_thread.detach();

    // Wait for the server to signal its initialization is done
    //sem_wait(&server_semaphore);
    
    // Launch the kernel
    client_thread_func<<<1,1>>>(h_sq_p, (LockFreeQueue<ResponseMessage> *)d_cq_p, 644);
    ASSERTDRV(hipStreamSynchronize(0));

    CUDA_ERRCHECK(hipDeviceSynchronize());
	CUDA_ERRCHECK(hipHostFree(h_sq_p));
    //CUDA_ERRCHECK(hipHostFree(h_cq_p));
    cudaGPUMemFree<LockFreeQueue<ResponseMessage>>(gpu_mm); //NEW
    delete(gpu_mm); //NEW
    return true;
}

int main() {
    PLIOPS_IDENTIFY_t identify = 0; //TODO guy check if I need a better identifier
    PLIOPS_DB_t plio_handle;
    if (!storelib_init(identify, plio_handle)){
        std::cout << "Storelib initialization failed. Exiting." << std::endl;       
        return 1;
    }

    if (!process_requests(plio_handle)) {
        std::cout << "Request processing failed. Exiting." << std::endl;       
        return 1;
    }

    if (!storelib_deinit(identify, plio_handle)){
        std::cout << "Storelib deinitialization failed. Exiting." << std::endl;       
        return 1;
    }
    return 0;
}